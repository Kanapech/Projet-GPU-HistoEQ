#include "hip/hip_runtime.h"
#include "cudaFunctions.hpp"
#include "utils/chronoGPU.hpp"

__global__
void rgb2hsvKernel(unsigned char* pixels, float* htab, float* stab, float* vtab, int width, int height){
    int x, y, offset, offsetHSV;
    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;

	offset = (y*width+x)*3;
    offsetHSV = (y*width+x);

    if(x < width && y < height){

        float r, g, b;
        r = pixels[offset] / 255.;
        g = pixels[offset+1] / 255.;
        b = pixels[offset+2] / 255.;

        float cmax = max(max(r, g), b);
        float cmin = min(min(r , g), b);
        float diff = cmax-cmin;

        if(diff == 0)
            htab[offsetHSV] = 0;
        else if(cmax == r)
            htab[offsetHSV] = (60 * ((g-b)/ diff) + 360) - ((int) ((60 * ((g-b)/ diff) + 360) / 360)) * 360; // fmod = numer - tquot * denom
        else if(cmax == g)
            htab[offsetHSV] = (60 * ((b-r)/ diff) + 120) - ((int) ((60 * ((b-r)/ diff) + 120) / 360)) * 360;
        else if(cmax == b)
            htab[offsetHSV] = (60 * ((r-g)/ diff) + 240) - ((int) ((60 * ((r-g)/ diff) + 240) / 360)) * 360;

        if(cmax == 0)
            stab[offsetHSV] = 0;
        else
            stab[offsetHSV] = 1 - (cmin/cmax);

        vtab[offsetHSV] = cmax;

    }
}

__host__
float rgb2hsvCompute(unsigned char* pixels, float* htab, float* stab, float* vtab, int width, int height){
    unsigned char* dev_pixels;
    float *dev_htab, *dev_stab, *dev_vtab;

    int size = width*height;
    // Allocate memory on Device
		HANDLE_ERROR(hipMalloc(&dev_pixels, size*3*sizeof(unsigned char)));
		HANDLE_ERROR(hipMalloc(&dev_htab, size*sizeof(float)));
        HANDLE_ERROR(hipMalloc(&dev_stab, size*sizeof(float)));
        HANDLE_ERROR(hipMalloc(&dev_vtab, size*sizeof(float)));
    
    // Copy from Host to Device
		HANDLE_ERROR(hipMemcpy(dev_pixels, pixels, size*3*sizeof(unsigned char), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_htab, htab, size*sizeof(float), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(dev_stab, stab, size*sizeof(float), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(dev_vtab, vtab, size*sizeof(float), hipMemcpyHostToDevice));

    //Kernel settings
        dim3 blockDim(32, 32);
        dim3 gridDim( width / blockDim.x, height / blockDim.y); 
        
    ChronoGPU chr;
	chr.start();

	// Launch kernel
		rgb2hsvKernel<<<gridDim, blockDim>>>(dev_pixels, dev_htab, dev_stab, dev_vtab, width, height);
	
	chr.stop();

	// Copy from Device to Host
		HANDLE_ERROR(hipMemcpy(pixels, dev_pixels, size*3*sizeof(unsigned char), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(htab, dev_htab, size*sizeof(float), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(stab, dev_stab, size*sizeof(float), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(vtab, dev_vtab, size*sizeof(float), hipMemcpyDeviceToHost));

	// Free memory on Device
		HANDLE_ERROR(hipFree(dev_pixels));
		HANDLE_ERROR(hipFree(dev_htab));
        HANDLE_ERROR(hipFree(dev_stab));
        HANDLE_ERROR(hipFree(dev_vtab));

	return chr.elapsedTime();
}