#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstring>

#include "image.hpp"
#include "seqFunctions.hpp"
#include "cudaFunctions.hpp"

using namespace std;

bool compare_float(float x, float y, float epsilon = 0.01f){
   if(fabs(x - y) < epsilon)
      return true; //they are same
      return false; //they are not same
}

int main(int argc, char **argv){

    cout << "======= Sequential on CPU ==========" << endl << endl;
    float timeCPU;
    Image img;
    img.load("img/Chateau.png");

    int size = img._height*img._width;
    float htab[size], stab[size], vtab[size];
        timeCPU = rgb2hsvCPU(img._pixels, htab, stab, vtab, img._width, img._height);
    cout << "rgb2hsvCPU : " << timeCPU << "ms" << endl;


    unsigned char* pixels = (unsigned char*) malloc(size*3*sizeof(unsigned char));
        timeCPU = hsv2rgbCPU(htab, stab, vtab, pixels, img._width, img._height);
    cout << "hsv2rgbCPU : " << timeCPU << "ms" << endl;

    Image im(img._width, img._height, 3);
    im._pixels = pixels;
    im.save("outputCPU.png");

    int* hist = (int*) calloc(256, sizeof(int));
        timeCPU = histogramCPU(vtab, hist, img._width, img._height);
    cout << "histoCPU : " << timeCPU << "ms" << endl;

    int* repart = (int*) calloc(256, sizeof(int));
        timeCPU = repartCPU(hist, repart);
    cout << "repartCPU : " << timeCPU << "ms" << endl;

    float* eqVtab = (float*) calloc(size, sizeof(float));
        timeCPU = equalizationCPU(repart, vtab, eqVtab, size);
    cout << "equalizationCPU : " << timeCPU << "ms" << endl;

    unsigned char* eqPixels = (unsigned char*) malloc(size*3*sizeof(unsigned char));
    hsv2rgbCPU(htab, stab, eqVtab, eqPixels, img._width, img._height);
    im._pixels = eqPixels;
    im.save("outputEQ.png");

    cout << endl;

    cout << "======= Parallel on GPU ==========" << endl << endl;

    float timeGPU;
    size = img._height*img._width;
    float *htabGPU = new float[size]; 
    float *stabGPU = new float[size];
    float *vtabGPU = new float[size];
        timeGPU = rgb2hsvCompute(pixels, htabGPU, stabGPU, vtabGPU, img._width, img._height);
    cout << "rgb2hsvGPU : " << timeGPU << "ms" << endl;

    int i=0;
    while(compare_float(htab[i], htabGPU[i]) && i < size)
        i++;
    cout << i << endl;
    cout << htab[i] << " " << htabGPU[i] << endl;

    unsigned char* pixelsGPU = (unsigned char*) malloc(size*3*sizeof(unsigned char));
    timeGPU = hsv2rgbCompute(htabGPU, stabGPU, vtabGPU, pixelsGPU, img._width, img._height);
    cout << "hsv2rgbGPU : " << timeGPU << "ms" << endl;
    im._pixels = pixelsGPU;
    im.save("outputGPU.png");

    i=0;
    while(pixels[i] == pixelsGPU[i] && i < size*3)
        i++;
    cout << i << endl;
    cout << pixels[i] << " " << pixelsGPU[i] << endl;

    int* histGPU = (int*) calloc(256, sizeof(int));
        timeGPU = histoCompute(vtabGPU, histGPU, img._width, img._height);
    cout << "histoGPU : " << timeGPU << "ms" << endl;

    /*i=0;
    while(i < 256){
        cout << hist[i]<< " "<< histGPU[i] << endl;
        i++;
    }*/

    i=0;
    while(hist[i] == histGPU[i] && i < 256)
        i++;
    cout << i << endl;
    cout << hist[i] << " " << histGPU[i] << endl;



    return 0;
}