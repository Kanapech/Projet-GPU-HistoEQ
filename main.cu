#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstring>

#include "image.hpp"
#include "seqFunctions.hpp"
#include "cudaFunctions.hpp"

using namespace std;

bool compare_float(float x, float y, float epsilon = 0.01f){
   if(fabs(x - y) < epsilon)
      return true; //they are same
      return false; //they are not same
}

int main(int argc, char **argv){

    cout << "======= Sequential on CPU ==========" << endl << endl;
    float timeCPU;
    float finalTimeCPU = 0;
    Image img;
    img.load("img/Chateau.png");

    int size = img._height*img._width;
    float *htab = new float[size]; 
    float *stab = new float[size];
    float *vtab = new float[size];
        timeCPU = rgb2hsvCPU(img._pixels, htab, stab, vtab, img._width, img._height);
    finalTimeCPU += timeCPU;
    cout << "rgb2hsvCPU : " << timeCPU << "ms" << endl;


    unsigned char* pixels = (unsigned char*) malloc(size*3*sizeof(unsigned char));
        timeCPU = hsv2rgbCPU(htab, stab, vtab, pixels, img._width, img._height);
    finalTimeCPU += timeCPU;
    cout << "hsv2rgbCPU : " << timeCPU << "ms" << endl;

    Image im(img._width, img._height, 3);
    im._pixels = pixels;
    im.save("outputCPU.png");

    int* hist = (int*) calloc(256, sizeof(int));
        timeCPU = histogramCPU(vtab, hist, img._width, img._height);
    finalTimeCPU += timeCPU;
    cout << "histoCPU : " << timeCPU << "ms" << endl;

    int* repart = (int*) calloc(256, sizeof(int));
        timeCPU = repartCPU(hist, repart);
    finalTimeCPU += timeCPU;
    cout << "repartCPU : " << timeCPU << "ms" << endl;

    float* eqVtab = (float*) calloc(size, sizeof(float));
        timeCPU = equalizationCPU(repart, vtab, eqVtab, img._width, img._height);
    finalTimeCPU += timeCPU;
    cout << "equalizationCPU : " << timeCPU << "ms" << endl;

    unsigned char* eqPixels = (unsigned char*) malloc(size*3*sizeof(unsigned char));
    hsv2rgbCPU(htab, stab, eqVtab, eqPixels, img._width, img._height);
    im._pixels = eqPixels;
    im.save("outputEQCPU.png");

    cout << endl;

    cout << "======= Parallel on GPU ==========" << endl << endl;

    float timeGPU;
    float finalTimeGPU = 0;
    size = img._height*img._width;

    // RGB2HSV //
    float *htabGPU = new float[size]; 
    float *stabGPU = new float[size];
    float *vtabGPU = new float[size];
        timeGPU = rgb2hsvCompute(pixels, htabGPU, stabGPU, vtabGPU, img._width, img._height);
    finalTimeGPU += timeGPU;
    cout << "rgb2hsvGPU : " << timeGPU << "ms" << endl;

    /*int i=0;
    while(compare_float(htab[i], htabGPU[i]) && i < size)
        i++;
    cout << i << endl;
    cout << htab[i] << " " << htabGPU[i] << endl;*/

    // HSV2RGB //
    unsigned char* pixelsGPU = (unsigned char*) malloc(size*3*sizeof(unsigned char));
        timeGPU = hsv2rgbCompute(htabGPU, stabGPU, vtabGPU, pixelsGPU, img._width, img._height);
    finalTimeGPU += timeGPU;
    cout << "hsv2rgbGPU : " << timeGPU << "ms" << endl;
    im._pixels = pixelsGPU;
    im.save("outputGPU.png");

    /*i=0;
    while(pixels[i] == pixelsGPU[i] && i < size*3)
        i++;
    cout << i << endl;
    cout << pixels[i] << " " << pixelsGPU[i] << endl;*/

    // histogram //
    int* histGPU = (int*) malloc(256*sizeof(int));
        timeGPU = histoCompute(vtabGPU, histGPU, img._width, img._height);
    finalTimeGPU += timeGPU;
    cout << "histoGPU : " << timeGPU << "ms" << endl;

    /*int i=0;
    while(hist[i] == histGPU[i] && i < 256)
        i++;
    cout << i << endl;
    cout << hist[i] << " " << histGPU[i] << endl;*/

    // repart //
    int* repartGPU = (int*) malloc(256*sizeof(int));
        timeGPU = repartCompute(histGPU, repartGPU);
    finalTimeGPU += timeGPU;
    cout << "repartGPU : " << timeGPU << "ms" << endl;

    /*i=0;
    while(repart[i] == repartGPU[i] && i < 256)
        i++;
    cout << i << endl;
    cout << repart[i] << " " << repartGPU[i] << endl;*/

    // Equalization //
    float* eqVtabGPU = (float*) malloc(size*sizeof(float));
        timeGPU = equalizationCompute(repartGPU, vtabGPU, eqVtabGPU, img._width, img._height);
    finalTimeGPU += timeGPU;
    cout << "equalizationGPU : " << timeGPU << "ms" << endl;

    unsigned char* eqPixelsGPU = (unsigned char*) malloc(size*3*sizeof(unsigned char));
    hsv2rgbCompute(htabGPU, stabGPU, eqVtabGPU, eqPixelsGPU, img._width, img._height);
    im._pixels = eqPixelsGPU;
    im.save("outputEQGPU.png");

    cout << endl << "======= Time Comparison ==========" << endl << endl;

    cout << "Final time CPU : " << finalTimeCPU << "ms" << endl;
    cout << "Final time GPU : " << finalTimeGPU << "ms" << endl;

    return 0;
}